﻿#include <hip/hip_runtime.h>
#include <>

#include <stdio.h>
#include <stdlib.h>

#include "bitmap.h"
#include "cannyEdgeDectection.h"
#include "common.h"
#include "rgbManip.h"

#define IN_IMAGE_NAME "mhwWallpaper4k.bmp"
//#define IN_IMAGE_NAME "lizard.bmp"

int main()
{
	unsigned char* fullBytesImageBuffer;
	int width, height;

	readBitmapImage(&fullBytesImageBuffer, &width, &height, IN_IMAGE_NAME);

	unsigned char* dev_fullBytesImageBuffer;
	CUDA_HANDLE_ERROR(hipMalloc(&dev_fullBytesImageBuffer, sizeof(unsigned char) * width * height * 3), "Cuda malloc");
	CUDA_HANDLE_ERROR(hipMemcpy(dev_fullBytesImageBuffer, fullBytesImageBuffer, sizeof(unsigned char) * width * height * 3, hipMemcpyHostToDevice), "Cuda memcpy to dev_fullBytesImageBuffer");
	
	float gaussianKernel5x5[5 * 5];
	computeGaussianKernel(gaussianKernel5x5, 5, 1);
	copyGaussianKernelToConstMem(gaussianKernel5x5);

	unsigned char* dev_grayBytesImageBuffer;
	unsigned char* dev_grayBytesOutCannyDetectionBuffer;
	CUDA_HANDLE_ERROR(hipMalloc(&dev_grayBytesImageBuffer, sizeof(unsigned char) * width * height), "Cuda malloc gray bytes buffer");
	CUDA_HANDLE_ERROR(hipMalloc(&dev_grayBytesOutCannyDetectionBuffer, sizeof(unsigned char) * width * height), "Cuda malloc gray bytes canny out buffer");
	cuda_rgbToGrayscale<<<128, 128>>>(dev_fullBytesImageBuffer, width, height, dev_grayBytesImageBuffer);

	cannyEdgeDetection(dev_grayBytesImageBuffer, width, height, dev_grayBytesOutCannyDetectionBuffer);

	cuda_grayBytesToGray3Bytes<<<128, 128>>>(dev_grayBytesOutCannyDetectionBuffer, width, height, dev_fullBytesImageBuffer);
	CUDA_HANDLE_ERROR(hipMemcpy(fullBytesImageBuffer, dev_fullBytesImageBuffer, sizeof(unsigned char) * width * height * 3, hipMemcpyDeviceToHost), "Memcpy device to host fullImageBytesBuffer");

	char outName[256];
	sprintf(outName, "out%s", IN_IMAGE_NAME);
	generateBitmapImage(fullBytesImageBuffer, height, width, outName);

	char startString[256];
	sprintf(startString, "start %s", outName);
	system(startString);
}

//int main()
//{
//	unsigned char** imageBytes;
//	unsigned char* dev_inputImageBytes;
//
//	int imageWidth, imageHeight;
//	readBitmapImage2D(&imageBytes, &imageWidth, &imageHeight, IN_IMAGE_NAME);
//	BGRToRGB2D(imageBytes, imageWidth, imageHeight);//TODO, this is unecessary because we can just treat the bytes in the BGR order in the code
//
//	unsigned char* imageGrayScale = (unsigned char*)malloc(sizeof(unsigned char) * imageWidth * imageHeight);
//	unsigned char* dev_cannyEdgeResult;
//
//	hipArray_t dev_imageGrayScaleArray;
//	hipChannelFormatDesc desc = hipCreateChannelDesc<unsigned char>();
//	CUDA_HANDLE_ERROR(hipMallocArray(&dev_imageGrayScaleArray, &desc, imageWidth, imageHeight, 0), "Cuda malloc array");
//
//	CUDA_HANDLE_ERROR(hipMalloc(&dev_inputImageBytes, sizeof(unsigned char) * imageWidth * imageHeight * 3), "Cuda malloc inputImageBytes");
//	CUDA_HANDLE_ERROR(hipMalloc(&dev_cannyEdgeResult, sizeof(unsigned char) * imageWidth * imageHeight), "Cuda malloc dev_cannyEdgeResult");
//
//	CUDA_HANDLE_ERROR(hipMemcpy(dev_inputImageBytes, imageBytes, sizeof(unsigned char) * imageWidth * imageHeight * 3, hipMemcpyHostToDevice), "Cuda memcpy inputImageBytes");
//
//	float gaussianKernel5x5[5 * 5];
//
//	computeGaussianKernel(gaussianKernel5x5, 5, 1);
//	copyGaussianKernelToConstMem(gaussianKernel5x5);
//
//	rgbToGrayscale<<<128, 128>>>(dev_inputImageBytes, imageWidth, imageHeight, dev_imageGrayScale);
//	CUDA_HANDLE_ERROR(hipFree(dev_inputImageBytes), "Cuda free dev_inputImagesBytes");
//	bindImageToTextureMemory(dev_imageGrayScaleArray, imageWidth, imageHeight);
//
//	hipEvent_t start, stop;
//	hipEventCreate(&start);
//	hipEventCreate(&stop);
//
//	hipEventRecord(start);
//	cannyEdgeDectection<<<128, 128>>>(dev_cannyEdgeResult, imageWidth, imageHeight);
//	hipEventRecord(stop);
//	hipEventSynchronize(stop);
//
//	float cannyEdgeDectectionDuration = 0;
//	hipEventElapsedTime(&cannyEdgeDectectionDuration, start, stop);
//	printf("Canny edge dectection time: %.3fms\n", cannyEdgeDectectionDuration);
//
//	CUDA_HANDLE_ERROR(hipMemcpy(imageGrayScale, dev_cannyEdgeResult, sizeof(unsigned char) * imageWidth * imageHeight, hipMemcpyDeviceToHost), "Cuda memcpy dev_imageGrayScale to imageBytes");
//
//	grayscaleToGrayscale3Bytes2D(imageGrayScale, imageWidth, imageHeight, imageBytes);
//	//BGRToRGB(imageBytes, imageWidth, imageHeight);//TODO, this is unecessary because we can just treat the bytes in the BGR order in the code
//
//	char imageOutName[256];
//	sprintf(imageOutName, "out%s", IN_IMAGE_NAME);
//
//	generateBitmapImage(imageBytes, imageHeight, imageWidth, imageOutName);
//
//	printf("\nDone.\n");
//}
