#include <hip/hip_runtime.h>


#include <stdio.h>

__global__ void cuda_rgbToGrayscale(unsigned char* input, int width, int height, unsigned char* output)
{
	int offset = threadIdx.x + blockIdx.x * blockDim.x;

	while (offset < width * height)
	{
		unsigned char red = input[offset * 3 + 0];
		unsigned char green = input[offset * 3 + 1];
		unsigned char blue = input[offset * 3 + 2];

		output[offset] = red * (float)0.299 + green * (float)0.587 + blue * (float)0.114;

		offset += gridDim.x * blockDim.x;
	}
}

__global__ void cuda_rgbToGrayscale3(unsigned char* input, int width, int height, unsigned char* output)
{
    int offset = threadIdx.x + blockIdx.x * blockDim.x;

    while (offset < width * height)
    {
        unsigned char red = input[offset * 3 + 0];
        unsigned char green = input[offset * 3 + 1];
        unsigned char blue = input[offset * 3 + 2];

        unsigned char grayValue = red * (float)0.299 + green * (float)0.587 + blue * (float)0.114;

        output[offset * 3 + 0] = grayValue;
        output[offset * 3 + 1] = grayValue;
        output[offset * 3 + 2] = grayValue;

        offset += gridDim.x * blockDim.x;
    }
}

//__global__ void cuda_rgbToGrayscale(unsigned char* input, int width, int height, unsigned char* output)
//{
//    int threadCount = gridDim.x * blockDim.x;
//    int stride = (width * height) / threadCount;//How many pixels the thread is going to process. This is pixelCount / threadCount
//    int remainder = width * height - stride * threadCount;//How many pixels are left to be processed because of pixelCount / threadCount not being an integer result
//
//    int threadId = threadIdx.x + blockIdx.x * blockDim.x;
//    int pixelOffset = threadId * stride;
//    int pixelOffset3 = threadId * stride * 3;
//
//    for (int i = 0; i < stride; i++)
//    {
//        unsigned char red = input[pixelOffset3 + i * 3 + 0];
//        unsigned char green = input[pixelOffset3 + i * 3 + 1];
//        unsigned char blue = input[pixelOffset3 + i * 3 + 2];
//
//        output[pixelOffset + i] = red * (float)0.299 + green * (float)0.587 + blue * (float)0.114;
//    }
//
//    if (threadId < remainder)
//    {
//        int offset = threadCount * stride + threadId;
//        int offset3 = threadCount * stride * 3 + threadId;
//
//        unsigned char red   = input[offset3 + 0];
//        unsigned char green = input[offset3 + 1];
//        unsigned char blue  = input[offset3 + 2];
//
//        output[offset] = red * (float)0.299 + green * (float)0.587 + blue * (float)0.114;
//    }
//}


__global__ void cuda_rgbToGrayscale2D(unsigned char** input, int width, int height, unsigned char** output)
{
	int offset = threadIdx.x + blockIdx.x * blockDim.x + threadIdx.y + blockIdx.y * blockDim.y * gridDim.x * blockDim.x * blockDim.y;

	while (offset < width * height)
	{
		int x = offset % width;
		int y = offset / width;

		unsigned char red   = input[y][x + 0];
		unsigned char green = input[y][x + 1];
		unsigned char blue  = input[y][x + 2];

		output[y][x] = red * 0.299 + green * 0.587 + blue * 0.114;

		offset += gridDim.x * blockDim.x;
	}
}

__host__ void BGRToRGB(unsigned char* imageBytes, int width, int height)
{
    for (int y = 0; y < height; y++)
    {
        for (int x = 0; x < width; x++)
        {
            int offset = (x + y * width) * 3;

            int currentBlue = imageBytes[offset + 0];
            int currentRed = imageBytes[offset + 2];

            imageBytes[offset + 0] = currentRed;
            imageBytes[offset + 2] = currentBlue;
        }
    }
}

__host__ void BGRToRGB2D(unsigned char** imageBytes, int width, int height)
{
    for (int y = 0; y < height; y++)
    {
        for (int x = 0; x < width; x++)
        {
            int currentBlue = imageBytes[y][x];
            int currentRed = imageBytes[y][x];

            imageBytes[y][x] = currentRed;
            imageBytes[y][x] = currentBlue;
        }
    }
}

__global__ void cuda_grayBytesToGray3Bytes(unsigned char* gray1ByteInput, int width, int height, unsigned char* gray3BytesOutput)
{
    int offset = threadIdx.x + blockIdx.x * blockDim.x;

    while (offset < width * height)
    {
        gray3BytesOutput[offset * 3 + 0] = gray1ByteInput[offset];
        gray3BytesOutput[offset * 3 + 1] = gray1ByteInput[offset];
        gray3BytesOutput[offset * 3 + 2] = gray1ByteInput[offset];

        offset += blockDim.x * gridDim.x;
    }
}

__host__ void grayscaleToGrayscale3Bytes(unsigned char* grayscaleInput, int width, int height, unsigned char* grayscale3Output)
{
    for (int y = 0; y < height; y++)
    {
        for (int x = 0; x < width; x++)
        {
            int index = y * width + x;

            grayscale3Output[index * 3 + 0] = grayscaleInput[index];
            grayscale3Output[index * 3 + 1] = grayscaleInput[index];
            grayscale3Output[index * 3 + 2] = grayscaleInput[index];
        }
    }
}

__host__ void grayscaleToGrayscale3Bytes2D(unsigned char** grayscaleInput, int width, int height, unsigned char** grayscale3Output)
{
    for (int y = 0; y < height; y++)
    {
        for (int x = 0; x < width; x++)
        {
            grayscale3Output[y][x + 0] = grayscaleInput[y][x];
            grayscale3Output[y][x + 1] = grayscaleInput[y][x];
            grayscale3Output[y][x + 2] = grayscaleInput[y][x];
        }
    }
}
