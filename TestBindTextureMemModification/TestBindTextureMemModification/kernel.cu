#include "hip/hip_runtime.h"
﻿#ifndef __HIPCC__
#define __HIPCC__
#endif

#include <hip/hip_runtime.h>
#include <>

#include <stdio.h>
#include <stdlib.h>

texture<int> textureArray;

__global__ void displayArray(int* arrayInt)
{
	int offset = threadIdx.x;

	printf("[%d]: %d\n", offset, arrayInt[offset]);
}

__global__ void displayTexture()
{
	int offset = threadIdx.x;

	printf("[%d]: %d\n", offset, tex1Dfetch(textureArray, offset));

	if(offset == 0)
		printf("\n\n");
}

__global__ void incrementArray(int* arrayInt)
{
	int offset = threadIdx.x;

	arrayInt[offset]++;
}

int main()
{
	int* arrayIntDevice;
	int* arrayInt = (int*)malloc(sizeof(int) * 5);
	for (int i = 0; i < 5; i++)
	{
		arrayInt[i] = i + 1;
		printf("%d\n", arrayInt[i]);
	}

	if(hipMalloc(&arrayIntDevice, sizeof(int) * 5) != 0)
		exit(-1);
	if (hipMemcpy(arrayIntDevice, arrayInt, sizeof(int) * 5, hipMemcpyHostToDevice) != 0)
		exit(-2);

	incrementArray <<<1, 5>>> (arrayIntDevice);

	hipChannelFormatDesc desc = hipCreateChannelDesc<int>();
	if (hipBindTexture(NULL, &textureArray, arrayIntDevice, &desc, sizeof(int) * 5) != 0)
		exit(-3);

	displayTexture << <1, 5 >> > ();

	incrementArray << <1, 5 >> > (arrayIntDevice);

	displayTexture << <1, 5 >> > ();
}
