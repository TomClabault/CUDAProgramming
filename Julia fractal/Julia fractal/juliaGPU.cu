#include "hip/hip_runtime.h"
﻿/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and
 * proprietary rights in and to this software and related documentation.
 * Any use, reproduction, disclosure, or distribution of this software
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA)
 * associated with this source code for terms and conditions that govern
 * your use of this NVIDIA software.
 *
 */

#include "benchOptions.h"

#include ""
#include "cudart_platform.h"
#include "hip/hip_runtime.h"

#include <math.h>
#include <time.h>
#include <stdlib.h>
#include <stdio.h>

struct hipComplex 
{
    float   r;
    float   i;

    __device__ hipComplex(float a, float b) : r(a), i(b) {}

    __device__ float magnitude2(void) 
    { 
        return r * r + i * i; 
    }

    __device__ hipComplex operator*(const hipComplex& a) 
    {
        return hipComplex(r * a.r - i * a.i, i * a.r + r * a.i);
    }

    __device__  hipComplex operator+(const hipComplex& a)
    {
        return hipComplex(r + a.r, i + a.i);
    }
};

__device__  int juliaGPU(int x, int y)
{
    float jx = SCALE * (float)(WIDTH / 2 - x) / (WIDTH / 2);
    float jy = SCALE * (float)(HEIGHT / 2 - y) / (HEIGHT / 2);

    hipComplex c(-0.8, 0.156);
    hipComplex a(jx, jy);

    int i = 0;
    for (i = 0; i < MAX_JULIA_ITERATION; i++) {
        a = a * a + c;
        if (a.magnitude2() > 1000)
            return 0;
    }

    return 1;
}

__global__ void kernelGPU(unsigned char* ptr) 
{
    const int rainbowColors[6][3] =
    {
        {255, 0, 0},   //Red
        {255, 255, 0}, //Yellow
        {0, 255, 0},   //Green
        {0, 255, 255}, //Cyan
        {0, 0, 255},   //Blue
        {255, 0, 255}, //Magenta
    };

    //TODO calculer x et Y d'abord et ensuite thread id, ça évitera le modulo et la div dans le while
    int threadId = gridDim.x * blockDim.x * blockDim.y * blockIdx.y + blockIdx.x * blockDim.x * blockDim.y + threadIdx.x + threadIdx.y * blockDim.x;

    while(threadId < WIDTH * HEIGHT)
    {
        int x = threadId % WIDTH;
        int y = threadId / WIDTH;

        int juliaValue = juliaGPU(x, y);

        float RGBpercent = (float)x / (WIDTH - 1);
        int firstColorIndex = (RGBpercent != 0) * ((int)ceil(RGBpercent * 6) - 1);

        int color[3] = { 0, 0, 0 };
        int secondColorIndex = (firstColorIndex + 1) % 6;

        float lowerBound = ((float)firstColorIndex / 6) * (firstColorIndex != 0);
        float upperBound = 1 * (firstColorIndex + 1 == 6) + ((float)secondColorIndex / 6) * (firstColorIndex + 1 != 6);//TODO remplacer par lower bound + 1/6

        float lerpPercent = 1.0 / ((upperBound - lowerBound) / (RGBpercent - lowerBound));

        color[0] = rainbowColors[firstColorIndex][0] * (1 - lerpPercent) + rainbowColors[secondColorIndex][0] * lerpPercent;
        color[1] = rainbowColors[firstColorIndex][1] * (1 - lerpPercent) + rainbowColors[secondColorIndex][1] * lerpPercent;
        color[2] = rainbowColors[firstColorIndex][2] * (1 - lerpPercent) + rainbowColors[secondColorIndex][2] * lerpPercent;

        ptr[threadId * 3 + 0] = color[0] * juliaValue;
        ptr[threadId * 3 + 1] = color[1] * juliaValue;
        ptr[threadId * 3 + 2] = color[2] * juliaValue;

        threadId += gridDim.x * gridDim.y * blockDim.x * blockDim.y;
    }
}
